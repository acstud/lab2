#include "hip/hip_runtime.h"
// Copyright 2018 Delft University of Technology
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>

#include "../baseline/water.hpp"

#include "water_cuda.hpp"

// An example CUDA kernel
__global__ void mul(float *a, float *b, float *c, int n) {
  for (int i = 0; i < n; i++) {
    c[i] = a[i] * b[i];
  }
}

std::shared_ptr<Image> runWaterEffectCUDA(const Image *src, const WaterEffectOptions *options) {
  /* REPLACE THIS CODE WITH YOUR OWN WATER EFFECT PIPELINE */

  std::cout << "Hello world compiled by the NVCC compiler!" << std::endl;

  float *a, *b, *c;

  // Allocate some CUDA unified memory
  hipMallocManaged(&a, 4 * sizeof(float));
  hipMallocManaged(&b, 4 * sizeof(float));
  hipMallocManaged(&c, 4 * sizeof(float));

  // Fill some dummy data
  for (int i = 0; i < 4; i++) {
    a[i] = 1.337f;
    b[i] = (float) i;
    c[i] = 0.0f;
  }

  // Start a kernel
  mul<<<1,1>>> (a, b, c, 4);

  // Wait for completion
  hipDeviceSynchronize();

  // Print the output
  for (int i = 0; i < 4; i++) {
    std::cout << a[i] << " * " << b[i] << " = " << c[i] << std::endl;
  }

  // Free the allocations
  hipFree(a);
  hipFree(b);
  hipFree(c);

  return nullptr;
  /* REPLACE THIS CODE WITH YOUR OWN WATER EFFECT PIPELINE */
}